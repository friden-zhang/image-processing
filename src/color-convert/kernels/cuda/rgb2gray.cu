#include "hip/hip_runtime.h"
#include <cub/block/block_reduce.cuh>


#include <cuda/atomic>

#include <cstdio>

#include "rgb2gray.cuh"

namespace image_processing {

namespace color_convert {

namespace kernels {

namespace detail {

// TODO: use cudax::span<T> (in cccl)
// see https://github.com/NVIDIA/cccl/tree/main/examples/cudax/vector_add
__global__ void rgb_packed_2_gray_kernel(const unsigned char *input,
                                         unsigned char *output, int width,
                                         int height) {

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    int index = (y * width + x) * 3;
    unsigned char r = input[index];
    unsigned char g = input[index + 1];
    unsigned char b = input[index + 2];

    output[y * width + x] =
        static_cast<unsigned char>(0.299f * r + 0.587f * g + 0.114f * b);
  }
}
} // namespace detail

bool launch_rgb_packed_2_gray_cuda(const unsigned char *input,
                                   unsigned char *output, int width,
                                   int height) {

  dim3 blockSize(32, 32);
  dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                (height + blockSize.y - 1) / blockSize.y);
  detail::rgb_packed_2_gray_kernel<<<gridSize, blockSize>>>(input, output,
                                                            width, height);
  // TODO: hanle error
  hipDeviceSynchronize();
  return true;
}
namespace detail {
// TODO: use cudax::span<T> (in cccl)
// see https://github.com/NVIDIA/cccl/tree/main/examples/cudax/vector_add
__global__ void rgb_planar_2_gray_kernel(const unsigned char *input,
                                         unsigned char *output, int width,
                                         int height) {

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = (y * width + x);
  int planar_size = width * height;

  if (x < width && y < height) {
    unsigned char r = input[index];
    unsigned char g = input[index + planar_size];
    unsigned char b = input[index + 2 * planar_size];

    output[y * width + x] =
        static_cast<unsigned char>(0.299f * r + 0.587f * g + 0.114f * b);
  }
}

} // namespace detail
bool launch_rgb_planar_2_gray_cuda(const unsigned char *input,
                                   unsigned char *output, int width,
                                   int height) {

  dim3 blockSize(32, 32);
  dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                (height + blockSize.y - 1) / blockSize.y);
  detail::rgb_planar_2_gray_kernel<<<gridSize, blockSize>>>(input, output,
                                                            width, height);
  // TODO: hanle error
  hipDeviceSynchronize();
  return true;
}

} // namespace kernels
} // namespace color_convert
} // namespace image_processing