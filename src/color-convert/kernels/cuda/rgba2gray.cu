#include "hip/hip_runtime.h"
#include <cub/block/block_reduce.cuh>

#include <thrust/device_vector.h>

#include <cuda/atomic>

#include <cstdio>

#include "rgba2gray.cuh"

namespace image_processing {

namespace color_convert {

namespace kernels {

namespace detail {

// TODO: use cudax::span<T> (in cccl)
// see https://github.com/NVIDIA/cccl/tree/main/examples/cudax/vector_add
__global__ void rgba_packed_2_gray_kernel(const unsigned char *input,
                                          unsigned char *output, int width,
                                          int height) {

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = (y * width + x) * 4;

  if (x < width && y < height) {
    unsigned char r = input[index];
    unsigned char g = input[index + 1];
    unsigned char b = input[index + 2];

    output[y * width + x] =
        static_cast<unsigned char>(0.299f * r + 0.587f * g + 0.114f * b);
  }
}
} // namespace detail

bool rgba_packed_2_gray_cuda(const unsigned char *input, unsigned char *output,
                             int width, int height) {

  // use device_trust vecrtor
  thrust::device_vector<unsigned char> input_dev(input,
                                                 input + 4 * width * height);
  thrust::device_vector<unsigned char> output_dev(width * height);

  unsigned char *input_dev_raw_ptr = thrust::raw_pointer_cast(input_dev.data());
  unsigned char *output_dev_raw_ptr =
      thrust::raw_pointer_cast(output_dev.data());
  dim3 blockSize(32, 32);
  dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                (height + blockSize.y - 1) / blockSize.y);
                
  detail::rgba_packed_2_gray_kernel<<<gridSize, blockSize>>>(input_dev_raw_ptr, output_dev_raw_ptr,
                                                             width, height);

  hipDeviceSynchronize();
  thrust::copy(output_dev.begin(), output_dev.end(), output);
  return true;
  return true;
}

namespace detail {
// TODO: use cudax::span<T> (in cccl)
// see https://github.com/NVIDIA/cccl/tree/main/examples/cudax/vector_add
__global__ void rgba_planar_2_gray_kernel(const unsigned char *input,
                                          unsigned char *output, int width,
                                          int height) {

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = (y * width + x);
  int planar_size = width * height;

  if (x < width && y < height) {
    unsigned char r = input[index];
    unsigned char g = input[index + planar_size];
    unsigned char b = input[index + 2 * planar_size];

    output[y * width + x] =
        static_cast<unsigned char>(0.299f * r + 0.587f * g + 0.114f * b);
  }
}
} // namespace detail

bool rgba_planar_2_gray_cuda(const unsigned char *input, unsigned char *output,
                             int width, int height) {

  // use device_trust vecrtor
  thrust::device_vector<unsigned char> input_dev(input,
                                                 input + 4 * width * height);
  thrust::device_vector<unsigned char> output_dev(width * height);

  unsigned char *input_dev_raw_ptr = thrust::raw_pointer_cast(input_dev.data());
  unsigned char *output_dev_raw_ptr =
      thrust::raw_pointer_cast(output_dev.data());

  dim3 blockSize(32, 32);
  dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                (height + blockSize.y - 1) / blockSize.y);
  detail::rgba_planar_2_gray_kernel<<<gridSize, blockSize>>>(
      input_dev_raw_ptr, output_dev_raw_ptr, width, height);

  hipDeviceSynchronize();
  thrust::copy(output_dev.begin(), output_dev.end(), output);
  return true;
  return true;
}

} // namespace kernels
} // namespace color_convert
} // namespace image_processing